// Multiplicação de matrizes em CUDA
// Disciplina: OPRP001 - Programação Paralela
// Prof.: Mauricio Pillon
// Aluno: Renato Tanaka

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

// Matriz Quadrada (nro_linhas = nro_colunas)
#define N 4 // Número de linhas 
            // Número de colunas

// GPU: Multiplicação das matrizes (a) e (b), resultado em (c)
__global__ void matMult (int *da, int *db, int *dc) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  dc[i*N+j] = 0;

  for(int k=0; k<N; k++)
    dc[i*N+j] += da[i*N+k] * db[k*N+j];
}

// GPU: Imprime índices na matriz 
__global__ void printIndex (void) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;

   printf ("[%d][%d]=%d\t(x)\t%d\t%d\t%d\t(y)\t%d\t%d\t%d\n",i,j,(i*N+j), threadIdx.x, blockIdx.x, blockDim.x,threadIdx.y, blockIdx.y, blockDim.y);
}
   

// GPU: Inicializa os vetores (a), (b) e (c) na Memória Global 
__global__ void dirtyMem (int *da, int *db, int *dc) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;

   da[i] = -1;
   db[i] = -2;
   dc[i] = -3;
}

// CPU: Inicializa os vetores (a) e (b)
__host__ void initvet(int *host_a, int *host_b) {
  for (int i=0; i < N; i++) {
    for (int j=0; j < N; j++) {
       host_b[i*N+j] = (i+j)+((N-1)*i);
       host_a[i*N+j] = (N*N)-host_b[i*N+j];
    }
  }
}

// CPU: Imprime matriz
__host__ void printMat (int *mat){

	for (int j =0; j < N; j++)
	printf("\t(%d)", j);
	printf("\n");
	for (int i=0; i < N; i++) {
		printf("(%d)", i);
		for (int j=0; j < N; j++){
			printf("\t%d", mat[i*N+j]);
		}
		printf("\n");
	}
}
 

// CPU: função principal 
int main(int argc, char const *argv[]) {
  int *a, *b, *c;
  int *dev_a, *dev_b, *dev_c;
  int size;

  // Alocação de matriz quadrada
  size = N * N * sizeof(int);

  // Alocação de memória no host
  hipHostMalloc((void **) &a, size, hipHostMallocDefault);
  hipHostMalloc((void **) &b, size, hipHostMallocDefault);
  hipHostMalloc((void **) &c, size, hipHostMallocDefault);
  
  // Alocação de memória na GPU para os vetores (a,b e c)
  hipMalloc ((void **) &dev_a, size);
  hipMalloc ((void **) &dev_b, size);
  hipMalloc ((void **) &dev_c, size);

  // Atribui valores iniciais aos vetores em GPU
  dirtyMem<<<N, N>>>(dev_a, dev_b, dev_c);

  // Cópia GPU para CPU 
  hipMemcpy (a, dev_a, size, hipMemcpyDeviceToHost);
  hipMemcpy (b, dev_b, size, hipMemcpyDeviceToHost);
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores Inicializados na GPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b);
  printf ("\t ### Matriz (c) ### \n");
  printMat(c);

  // Inicialização dos vetores (a) e (b) no host
  initvet(a,b);

  // Cópia dos vetores gerados em CPU p/ memória da GPU
  hipMemcpy (dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy (dev_b, b, size, hipMemcpyHostToDevice);

  // Número de blocos e threads p/ dimensões (x,y)
  dim3 dimBlock (1, 1); 
  dim3 dimThreads(N, N); 

  // Imprime as posições acessadas pelo dimBlock e dimThreads
  printIndex<<< dimBlock, dimThreads>>>();

  // Execução do kernel matMult em GPU
  matMult<<< dimBlock, dimThreads>>>(dev_a, dev_b, dev_c);
  hipDeviceSynchronize();

  // Cópia do vetor (c) da GPU (Memória Global) para CPU
  hipMemcpy (c, dev_c, size, hipMemcpyDeviceToHost);

  // Impressão na tela dos valores dos vetores
  printf ("\t ### Valores após processamento em GPU ###\n");
  printf ("\t ### Matriz (a) ### \n");
  printMat(a);
  printf ("\t ### Matriz (b) ### \n");
  printMat(b);
  printf ("\t ### Matriz (c) ### \n");
  printMat(c);

  // Libera a Memória Global (GPU)
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // Libera a Memória Global (CPU)
  hipHostFree(a);
  hipHostFree(b);
  hipHostFree(c);

  return 0;
}
